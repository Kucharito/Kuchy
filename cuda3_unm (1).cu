#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Manipulation with prepared image.
//
// ***********************************************************************

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_img.h"

__global__ void kernel_flip(CudaImg t_cuda_img, char axis){
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    int half_x = t_cuda_img.m_size.x / 2;
    int half_y = t_cuda_img.m_size.y / 2;

    if (x >= t_cuda_img.m_size.x) return;
    if (y >= t_cuda_img.m_size.y) return;

    if (axis == 'x'){
        if ((y + half_y) < t_cuda_img.m_size.y){
            uchar3 temp = t_cuda_img.at3(x, y + half_y);
            t_cuda_img.at3(x, y + half_y) = t_cuda_img.at3(x, y);
            t_cuda_img.at3(x, y) = temp;
        }
    }

    if (axis == 'y'){
        if ((x + half_x) < t_cuda_img.m_size.x){
            uchar3 temp = t_cuda_img.at3(x + half_x, y);
            t_cuda_img.at3(x + half_x, y) = t_cuda_img.at3(x, y);
            t_cuda_img.at3(x, y) = temp;
        }
    }

    return;
}

__global__ void kernel_rotate(CudaImg image, CudaImg rotate, char side ){
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x >= image.m_size.x) return;
    if (y >= image.m_size.y) return;

    if (side == '<'){
        rotate.at3(y, x) = image.at3(x, y);
    }
    
    if (side == '>'){
        rotate.at3(y, x) = image.at3(x, y);
    }

    return;
}

void cu_run_flip(CudaImg t_cuda_img, char axis){
    int block_size = 16;
    dim3 blockamount((t_cuda_img.m_size.x + block_size - 1) / block_size, (t_cuda_img.m_size.y + block_size - 1) / block_size);
    dim3 threadsperblock(block_size, block_size);
    kernel_flip<<<blockamount, threadsperblock>>>(t_cuda_img, axis);
    hipDeviceSynchronize();
    return;
}

void cu_run_rotate(CudaImg image, CudaImg rotate, char side){
    int block_size = 16;
    dim3 blockamount((image.m_size.x + block_size - 1) / block_size, (image.m_size.y + block_size - 1) / block_size);
    dim3 threadsperblock(block_size, block_size);
    kernel_rotate<<<blockamount, threadsperblock>>>(image, rotate, side);
    hipDeviceSynchronize();
    printf("%d %d // %d %d ", image.m_size.x, image.m_size.y, rotate.m_size.x, rotate.m_size.y);
    return;
}